#include "hip/hip_runtime.h"
// Copyright 2013 Francisco Souza. All rights reserved.
// Use of this source code is governed by a BSD-style
// license that can be found in the LICENSE file.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void
checkError(hipError_t err, int line)
{
	if(err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, line);
		exit(EXIT_FAILURE);
	}
}

__global__ void
vecAddKernel(float *a, float *b, float *c, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i < n) {
		c[i] = a[i] + b[i];
	}
}

__host__ void
vecAdd(float *ha, float *hb, float *hc, int n)
{
	hipError_t err;
	int size = n * sizeof(data);
	float *da, *db, *dc;
	err = hipMalloc((void **) &da, size);
	checkError(err, __LINE__);
	err = hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
	checkError(err, __LINE__);
	err = hipMalloc((void **) &db, size);
	checkError(err, __LINE__);
	err = hipMemcpy(db, hb, size, hipMemcpyHostToDevice);
	checkError(err, __LINE__);
	err = hipMalloc((void **) &dc, size);
	checkError(err, __LINE__);
	vecAddKernel<<<ceil(n/256.0),256>>>(da, db, dc, n);
	err = hipMemcpy(hc,dc, size, hipMemcpyDeviceToHost);
	checkError(err, __LINE__);
	err = hipFree(da);
	checkError(err, __LINE__);
	err = hipFree(db);
	checkError(err, __LINE__);
	err = hipFree(dc);
	checkError(err, __LINE__);
}
